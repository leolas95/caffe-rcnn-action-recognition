#include "caffe/FRCNN/frcnn_roi_data_layer.hpp"

namespace caffe {

namespace Frcnn {

template <typename Dtype>
void FrcnnRoiDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = this->prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data, Image Blob
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // top[1] is image_info , top[2] is gx_bpxes
    caffe_copy(3, batch->label_.gpu_data(), top[1]->mutable_gpu_data());
    // Reshape to loaded labels.
    vector<int> label_shape(batch->label_.shape());
    label_shape[0] = label_shape[0] - 1;
    top[2]->Reshape(label_shape);
    // Copy the labels.
    // First five is image_info
    caffe_copy(batch->label_.count() - 5, batch->label_.gpu_data() + 5,
        top[2]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  this->prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(FrcnnRoiDataLayer);

}  // namespace Frcnn

}  // namespace caffe
