#include "caffe/FRCNN/frcnn_roi_data_layer.hpp"

namespace caffe {

namespace Frcnn {

template <typename Dtype>
void FrcnnRoiDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = this->prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data, Image Blob
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(), top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // top[1] is image_info , top[2] is gx_bpxes
    caffe_copy(3, batch->label_.gpu_data(), top[1]->mutable_gpu_data());
    // Reshape to loaded labels.
    top[2]->Reshape(batch->label_.num()-1, batch->label_.channels(), batch->label_.height(), batch->label_.width());
    // Copy the labels.
    // First five is image_info
    caffe_copy(batch->label_.count() - 5, batch->label_.gpu_data() + 5, top[2]->mutable_gpu_data());

    // DEBUG INFORMATION
    DLOG(ERROR) << "[" << (this->phase_==TRAIN?"TRAIN":"TEST") << "] FrcnnRoiDataLayer Forward: " << image_database_[batch->label_.cpu_data()[3]] << " scale : " << top[1]->cpu_data()[2] << " [] GT_NUM : " << top[2]->num();
    DLOG(ERROR) << "FrcnnRoiDataLayer Forward: " << "height : " << top[1]->cpu_data()[0] << ", width : " << top[1]->cpu_data()[1] ;
    for (int i = 0; i < top[2]->num(); i++) {
      DLOG(ERROR) << "---------= " << top[2]->cpu_data()[5 * i + 0] << ", " << top[2]->cpu_data()[5 * i + 1] << ", " << top[2]->cpu_data()[5 * i + 2] << ", " << top[2]->cpu_data()[5 * i + 3] << "   . label : " << top[2]->cpu_data()[5 * i + 4];
    }
    DLOG(ERROR) << "FrcnnRoiDataLayer Forward >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> END";
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  this->prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(FrcnnRoiDataLayer);

}  // namespace Frcnn

}  // namespace caffe
